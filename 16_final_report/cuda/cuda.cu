
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <chrono>
#include <stdlib.h>
using namespace std;
#define M 32

__global__ void matrix(int N, float *A, float *B, float* C){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float s1[M][M+1];
    __shared__ float s2[M][M+1];

    float sum=0;
    for(int i =0;i<N/M;i++)
{
    s1[ty][tx]=A[(by*M+ty)*N+(i*M+tx)];
    s2[ty][tx]=B[(bx*M+tx)+(i*M+ty)*N];
    __syncthreads();

   for (int k=0;k<M;k++)
      sum+=s1[ty][k]*s2[k][tx];
    __syncthreads();

}
       C[(by*M+ty)*N+bx*M+tx]=sum;
}


int main(int argc, char** argv) {
  const int N = 256;
    vector<float> A(N*N);
    vector<float> B(N*N);
    vector<float> C(N*N, 0);

    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            A[N*i+j] = drand48();
            B[N*i+j] = drand48();
        }
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(float) * N * N);
    hipMalloc(&d_B, sizeof(float) * N * N);
    hipMalloc(&d_C, sizeof(float) * N * N);

    hipMemcpy(d_A, &A[0], sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, &B[0], sizeof(float) * N * N, hipMemcpyHostToDevice);
  
  auto tic = chrono::steady_clock::now();

  int GRID_SIZE=(N+M-1)/M;
  dim3 grid(GRID_SIZE,GRID_SIZE);
  dim3 block(M,M);
  matrix<<<grid,block>>>(N,d_A,d_B,d_C);
  hipDeviceSynchronize();

  auto toc = chrono::steady_clock::now();
  double time = chrono::duration<double>(toc-tic).count();
  
  hipMemcpy(&C[0], d_C, sizeof(float) * N * N, hipMemcpyDeviceToHost);

  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      for (int k=0; k<N; k++)
        C[N*i+j] -= A[N*i+k] * B[N*k+j];
  double err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(C[N*i+j]);

  printf("N    : %d\n",N);
  printf("total: %lf s (%lf GFlops)\n",time,2.*N*N*N/time/1e9);
  printf("error: %lf\n",err/N/N);
}
